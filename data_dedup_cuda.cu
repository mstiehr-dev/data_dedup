#include "hip/hip_runtime.h"
/* data_dedup_cuda.cu */

#include "data_dedup.h"
#include "data_dedup.cuh"



__global__ void searchKernel(void *entrySet, long *result, int entries) {
	// implementiert memcmp auf Basis von <long> Vergleichen 
	long idx = threadIdx.x + blockIdx.x * blockDim.x;
	const long *c1,*c2;
	char n, diff; // diff: der aktuelle Thread soll nicht öfter laufen, als nötig (auf gesamten Kernelaufruf nicht ausweitbar) 
	while(idx<entries) { // Threads werden recycled, siehe Inkrement am Fuß der Schleife
		diff = 0; // FALSE
		n=32/sizeof(long); // 4 Vergleiche 
		// Pointer jeweils auf den Anfang setzen 
		c1 = (long *)goldenHash;
		c2 = (long *)((journalentry *)entrySet)[idx].hash;
		while(n--) {
			if(*c1 != *c2) { // Abweichung
				diff = 1;
				break;
			}
			c1++;
			c2++;
		}
		if(!diff) { // treffer
			*result = idx; // Thread-Index ist die Nummer des Eintrags
			idx = entries; // dieser thread braucht nicht weitersuchen
		}
		idx += blockDim.x * gridDim.x; // aktueller index + (anzahl der Blöcke * Threads pro Block) 
	}
	return;
} 



__host__ long isHashInJournalGPU(char *hash, void *haystack, int stacksize) {
	CUDA_HANDLE_ERR( hipMemcpyToSymbol(HIP_SYMBOL(goldenHash), hash, 32) ); // die gesuchte Prüfsumme wird in den Cache der GPU gebracht 
	long result = -1L;
	searchKernel<<<blocks,threadsPerBlock>>>(haystack, &result, stacksize);
	return result;
}

__host__ void cudaCopyJournal(void *dev, void *host, off_t len) {
	CUDA_HANDLE_ERR( hipMalloc((void**)&dev, len) ); // GPU Speicher wird alloziert
	CUDA_HANDLE_ERR( hipMemcpy(dev, host, len, hipMemcpyHostToDevice) ); // Datentransfer von Host Speicher nach VRAM 
}

__host__ void cudaExtendHashStack(void *add, journalentry *entry) {
	CUDA_HANDLE_ERR( hipMemcpy(add, record, sizeof(journalentry), hipMemcpyHostToDevice) );
}