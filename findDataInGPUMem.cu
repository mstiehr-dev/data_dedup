#include "hip/hip_runtime.h"
/* findDataInGPUMem.cu */



#include "data_dedup.h" 
#include "unistd.h" //getopt

/* globale variablen: */
#define _haystack (10*1024)
int _blocks;
int _threads;
__constant__ journalentry findMe[1];
/* hier wird der gesuchte Hash gespeichert
 * __constant__ heißt, dass der Wert in einem Cache der GPU gehalten wird */




__device__ int compareHashes(/*const char *s1, */const char *s2, size_t n) {
	const char *c1=findMe[0].hash, *c2=s2;
	while(n--) {
		if(*c1!=*c2)
			return (*c2-*c1);
		c1++;
		c2++;
	}
	return 0;
}
__global__ void kernel(/*void *wantedEntry, */void *entrySet, int *resp, int entries) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	while(idx<_haystack) { // den wantedhash irgendwo cachen!!! 
		if(compareHashes( /*((journalentry *)wantedEntry)->hash, */((journalentry *)entrySet+idx)->hash,32) == 0 ) {
			// Treffer -> alle anderen können aufhören
			*resp = idx; // wird ständig überschrieben, weil kernel auch nach treffer nicht terminiert
			//asm("trap;"); // ekliger abbruch, führt zu fehlern
			return;
		}
		idx += blockDim.x * gridDim.x;
	}
	// wenn wir hier ankommen, gab es keinen Treffer -> Fehlercode -1
	//*resp = (-1);
	return;
} 




int main(int argc, char **argv) {
/* Vorbereitungen */
	// Command Line Arguments parsen: 
	int c;
	opterr = 0;
	while((c=getopt(argc, argv, "b:t:"))!=-1) { // : -> argument required
		switch(c) {
			//case 'h':	_haystack = atoi(optarg); break;
			case 'b':	if(optarg) _blocks   = atoi(optarg); break;
			case 't':	if(optarg) _threads  = atoi(optarg); break;
			default:
				printf("usage: %s -h <size of _haystack> -b <_blocks> -t <_threads per block>\n",argv[0]);
				exit(1);
		}
	}
	printf("%d | %d | %d \n", _haystack, _blocks, _threads);
	exit(0);






	srand(time(NULL));
	unsigned int treffer = randFloat() * _haystack; // Dieser Datensatz wird nachher im _haystack gesucht
	hipEvent_t start, stop; 
	float elapsedTime;
	// wieviel Speicher hat die GPU? 
	hipDeviceProp_t gpu; 
	CUDA_HANDLE_ERR( hipGetDeviceProperties(&gpu, 0) );
	size_t totalGPUMem = gpu.totalGlobalMem;
	if(_haystack*sizeof(journalentry)>=totalGPUMem) {
		char user[2]; // 1 Buchstabe + \n
		printf("+++ WARNING +++\n");
		printf("+++ Memory usage exceeds GPU capacity!\n");
		printf("+++ continue? (y/N)\n");
		printf(" > ");
		fgets(user, 2, stdin);
		if('Y'!=(*user&0x59)) {
			printf("exit");
			exit(1);
		}
	}
	// --- lokalen Speicher bereitstellen und initialisieren 
	journalentry * host_data = (journalentry *) malloc(_haystack*sizeof(journalentry));
	memset(host_data, 0, _haystack*sizeof(journalentry));
	int i;
	for(i=0; i<_haystack; i++) {
		(host_data+i)->block = LONG_MAX * randFloat();
		(host_data+i)->len = SHRT_MAX * randFloat();
		char *tString = randString(32);
		strncpy((host_data+i)->hash,tString,32);
		free(tString);
	}

	CUDA_HANDLE_ERR( hipEventCreate(&start)   );
	CUDA_HANDLE_ERR( hipEventCreate(&stop)    );
	CUDA_HANDLE_ERR( hipEventRecord(start, 0) );
	
	// datensätze auf GPU bringen 
	void * dev_data;
	CUDA_HANDLE_ERR( hipMalloc((void**)&dev_data, _haystack*sizeof(journalentry)) );
	CUDA_HANDLE_ERR( hipMemcpy(dev_data, host_data, _haystack*sizeof(journalentry), hipMemcpyHostToDevice) );
	
	
	//void * dev_wantedEntry; 
	CUDA_HANDLE_ERR( hipMemcpyToSymbol(HIP_SYMBOL(findMe), host_data+treffer, sizeof(journalentry)) );
	//memcpy(&findMe, host_data+treffer, sizeof(journalentry));
	printf("so we're looking for this hash: [%s]\n", host_data[treffer].hash);
	//CUDA_HANDLE_ERR( hipMalloc((void**)&dev_wantedEntry, sizeof(journalentry)) );
	//CUDA_HANDLE_ERR( hipMemcpy(dev_wantedEntry, &findMe, sizeof(journalentry), hipMemcpyHostToDevice) );
	
	// außerdem muss der Kernel irgendwo die Antwort speichern können: 
	int host_resp=-1; 
	int *dev_resp; 
	CUDA_HANDLE_ERR( hipMalloc((void**)&dev_resp, sizeof(int)) );
	CUDA_HANDLE_ERR( hipMemcpy(dev_resp, &host_resp, sizeof(int), hipMemcpyHostToDevice) );
	kernel<<<_blocks,_threads>>>(/*dev_wantedEntry, */dev_data, dev_resp, _haystack);
	CUDA_HANDLE_ERR( hipMemcpy(&host_resp, dev_resp, sizeof(int), hipMemcpyDeviceToHost) );
	CUDA_HANDLE_ERR( hipEventRecord(stop,0) );
	CUDA_HANDLE_ERR( hipEventSynchronize(stop) );
	CUDA_HANDLE_ERR( hipEventElapsedTime(&elapsedTime, start, stop) );
	if(host_resp>=0) 
		printf("got your hash in tupel #%d!\n",host_resp);
	else
		printf("sorry pal - return value is %d\n", host_resp);
	printf("### computation took %fms\n",elapsedTime);
	printf("### Using %d _threads in a (%dx%d) Grid\n", (_blocks*_threads), _blocks, _threads);
	printf("### _haystack: %d\n",_haystack);
	//CUDA_HANDLE_ERR( hipFree(dev_wantedEntry) );
	CUDA_HANDLE_ERR( hipFree(dev_data) );
	CUDA_HANDLE_ERR( hipFree(dev_resp) );
	CUDA_HANDLE_ERR( hipEventDestroy(start) );
	CUDA_HANDLE_ERR( hipEventDestroy(stop) );
	free(host_data);
	
	return 0;
}








char randChar() {
	//liefert ein zufälliges druckbares Zeichen 
	/* Druckbare Zeichen beginnen bei 32 (A) und enden bei 126 (~) -> Spanne von 94 */
	const char start = 'A';
	const char end   = '~';
	const char range = end-start;
	char  c = 32 + randFloat()*range; 
	return c;
}
char * randString(size_t n) {
	// liefert eine zufällige Zeichenkette 
	char *str = (char *)malloc(n*sizeof(char)+1);
	if(str==NULL) {
		perror("malloc() failed in randString()");
		exit(1);
	}
	str[n] = '\0'; // Stringende 
	while(n--)
		str[n] = randChar();
	return str;
}

float randFloat() { // liefert eine Zufallszahl zwischen 0 und 1 (inklusive) 
	return ((float)rand())/RAND_MAX;
}
